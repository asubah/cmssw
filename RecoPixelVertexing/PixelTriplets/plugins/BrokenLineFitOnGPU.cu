#include "BrokenLineFitOnGPU.h"
#include "HeterogeneousCore/CUDAUtilities/interface/device_unique_ptr.h"

template <typename TrackerTraits>
void HelixFitOnGPU<TrackerTraits>::launchBrokenLineKernels(const TrackingRecHitSoAConstView<TrackerTraits>& hv,
                                                           uint32_t hitsInFit,
                                                           uint32_t maxNumberOfTuples,
                                                           cms::LaunchConfigs const &kernelConfigs,
                                                           hipStream_t stream) {
  assert(tuples_);

  cms::LaunchConfig config = kernelConfigs.getConfig("kernel_BLFastFit");
  auto blockSize_BLFastFit = config.threads[0] > 0 ? config.threads[0] : 64;
  auto numberOfBlocks_BLFastFit = config.blocks[0] > 0 ? config.blocks[0] : ((maxNumberOfConcurrentFits_ + blockSize_BLFastFit - 1) / blockSize_BLFastFit);

  config = kernelConfigs.getConfig("kernel_BLFit");
  auto blockSize_BLFit = config.threads[0] > 0 ? config.threads[0] : 64;
  auto numberOfBlocks_BLFit = config.blocks[0] > 0 ? config.blocks[0] : ((maxNumberOfConcurrentFits_ + blockSize_BLFit - 1) / blockSize_BLFit);

  //  Fit internals
  auto tkidGPU =
      cms::cuda::make_device_unique<typename TrackerTraits::tindex_type[]>(maxNumberOfConcurrentFits_, stream);
  auto hitsGPU = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Matrix3xNd<6>) / sizeof(double), stream);
  auto hits_geGPU = cms::cuda::make_device_unique<float[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Matrix6xNf<6>) / sizeof(float), stream);
  auto fast_fit_resultsGPU = cms::cuda::make_device_unique<double[]>(
      maxNumberOfConcurrentFits_ * sizeof(riemannFit::Vector4d) / sizeof(double), stream);

  for (uint32_t offset = 0; offset < maxNumberOfTuples; offset += maxNumberOfConcurrentFits_) {
    // fit triplets

    kernel_BLFastFit<3, TrackerTraits><<<numberOfBlocks_BLFastFit, blockSize_BLFastFit, 0, stream>>>(tuples_,
                                                                                 tupleMultiplicity_,
                                                                                 hv,
                                                                                 tkidGPU.get(),
                                                                                 hitsGPU.get(),
                                                                                 hits_geGPU.get(),
                                                                                 fast_fit_resultsGPU.get(),
                                                                                 3,
                                                                                 3,
                                                                                 offset);
    cudaCheck(hipGetLastError());

    kernel_BLFit<3, TrackerTraits><<<numberOfBlocks_BLFit, blockSize_BLFit, 0, stream>>>(tupleMultiplicity_,
                                                                             bField_,
                                                                             outputSoa_,
                                                                             tkidGPU.get(),
                                                                             hitsGPU.get(),
                                                                             hits_geGPU.get(),
                                                                             fast_fit_resultsGPU.get());
    cudaCheck(hipGetLastError());

    if (fitNas4_) {
      // fit all as 4
      riemannFit::rolling_fits<4, TrackerTraits::maxHitsOnTrack, 1>([this,
                                                                     &hv,
                                                                     &tkidGPU,
                                                                     &hitsGPU,
                                                                     &hits_geGPU,
                                                                     &fast_fit_resultsGPU,
                                                                     &offset,
                                                                     &numberOfBlocks_BLFastFit,
                                                                     &numberOfBlocks_BLFit,
                                                                     &blockSize_BLFastFit,
                                                                     &blockSize_BLFit,
                                                                     &stream](auto i) {
        kernel_BLFastFit<4, TrackerTraits><<<numberOfBlocks_BLFastFit / 4, blockSize_BLFastFit, 0, stream>>>(tuples_,
                                                                                         tupleMultiplicity_,
                                                                                         hv,
                                                                                         tkidGPU.get(),
                                                                                         hitsGPU.get(),
                                                                                         hits_geGPU.get(),
                                                                                         fast_fit_resultsGPU.get(),
                                                                                         4,
                                                                                         4,
                                                                                         offset);

        cudaCheck(hipGetLastError());

        kernel_BLFit<4, TrackerTraits><<<numberOfBlocks_BLFit / 4, blockSize_BLFit, 0, stream>>>(tupleMultiplicity_,
                                                                                     bField_,
                                                                                     outputSoa_,
                                                                                     tkidGPU.get(),
                                                                                     hitsGPU.get(),
                                                                                     hits_geGPU.get(),
                                                                                     fast_fit_resultsGPU.get());

        cudaCheck(hipGetLastError());
      });

    } else {
      riemannFit::rolling_fits<4, TrackerTraits::maxHitsOnTrackForFullFit, 1>([this,
                                                                               &hv,
                                                                               &tkidGPU,
                                                                               &hitsGPU,
                                                                               &hits_geGPU,
                                                                               &fast_fit_resultsGPU,
                                                                               &offset,
                                                                               &numberOfBlocks_BLFastFit,
                                                                               &numberOfBlocks_BLFit,
                                                                               &blockSize_BLFastFit,
                                                                               &blockSize_BLFit,
                                                                               &stream](auto i) {
        kernel_BLFastFit<i, TrackerTraits><<<numberOfBlocks_BLFastFit / 4, blockSize_BLFastFit, 0, stream>>>(tuples_,
                                                                                         tupleMultiplicity_,
                                                                                         hv,
                                                                                         tkidGPU.get(),
                                                                                         hitsGPU.get(),
                                                                                         hits_geGPU.get(),
                                                                                         fast_fit_resultsGPU.get(),
                                                                                         i,
                                                                                         i,
                                                                                         offset);

        kernel_BLFit<i, TrackerTraits><<<8, blockSize_BLFit, 0, stream>>>(tupleMultiplicity_,
                                                                    bField_,
                                                                    outputSoa_,
                                                                    tkidGPU.get(),
                                                                    hitsGPU.get(),
                                                                    hits_geGPU.get(),
                                                                    fast_fit_resultsGPU.get());
      });

      static_assert(TrackerTraits::maxHitsOnTrackForFullFit < TrackerTraits::maxHitsOnTrack);

      //Fit all the rest using the maximum from previous call
      kernel_BLFastFit<TrackerTraits::maxHitsOnTrackForFullFit, TrackerTraits>
          <<<numberOfBlocks_BLFastFit/ 4, blockSize_BLFastFit, 0, stream>>>(tuples_,
                                                         tupleMultiplicity_,
                                                         hv,
                                                         tkidGPU.get(),
                                                         hitsGPU.get(),
                                                         hits_geGPU.get(),
                                                         fast_fit_resultsGPU.get(),
                                                         TrackerTraits::maxHitsOnTrackForFullFit,
                                                         TrackerTraits::maxHitsOnTrack - 1,
                                                         offset);

      kernel_BLFit<TrackerTraits::maxHitsOnTrackForFullFit, TrackerTraits>
          <<<8, blockSize_BLFit, 0, stream>>>(tupleMultiplicity_,
                                        bField_,
                                        outputSoa_,
                                        tkidGPU.get(),
                                        hitsGPU.get(),
                                        hits_geGPU.get(),
                                        fast_fit_resultsGPU.get());
    }

  }  // loop on concurrent fits
}

template class HelixFitOnGPU<pixelTopology::Phase1>;
template class HelixFitOnGPU<pixelTopology::Phase2>;
